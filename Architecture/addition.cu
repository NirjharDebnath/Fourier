#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <hipblas.h>

int main() {
    int n = 1 << 16;
    float alpha = 2.0f;

    std::vector<float> h_x(n), h_y(n);
    for (int i = 0; i < n; i++) {
        h_x[i] = static_cast<float>(i);
        h_y[i] = static_cast<float>(2 * i);
    }

    float *d_x, *d_y;
    hipMalloc((void**)&d_x, n * sizeof(float));
    hipMalloc((void**)&d_y, n * sizeof(float));

    hipMemcpy(d_x, h_x.data(), n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y.data(), n * sizeof(float), hipMemcpyHostToDevice);
    
    // cuBLAS handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Perform y = alpha*x + y
    hipblasSaxpy(handle, n, &alpha, d_x, 1, d_y, 1);

    // Copy back result
    hipMemcpy(h_y.data(), d_y, n * sizeof(float), hipMemcpyDeviceToHost);

    // Verify result
    bool success = true;
    for (int i = 0; i < n; i++) {
        if (h_y[i] != 2.0f * i + alpha * i) {
            std::cerr << "Mismatch at " << i << ": " << h_y[i] << std::endl;
            success = false;
            break;
        }
    }
    if (success) std::cout << "AXPY successful!" << std::endl;

    // Cleanup
    hipblasDestroy(handle);
    hipFree(d_x);
    hipFree(d_y);

    return 0;
}
